#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#define N 50
#define NumBlocks  (ThreadsPerBlock + (N*M-1))/ThreadsPerBlock
#define THREADS_PER_BLOCK 512   //size of block

__device__ void suddenDeath(double m[2][2], int n, int *attacker, int f1,int f2, char g1, char g2){
    for (int i = 0; i < n; ++i)
    {
        if(f1 == 0){
            printf("\nGroup %c is annihilated!\n\n",g1);
            printf("==================================\nGroup %c is the winner!\n==================================\n",g2);
            return;
        }else if(f2 == 0){
            printf("\nGroup %c is annihilated!\n\n",g2);
            printf("==================================\nGroup %c is the winner!\n==================================\n",g1);
            return;
        }

        if(i%2==0){
        *attacker = 0;
        }else{
            *attacker = 1;
        }

        switch(*attacker){
            case 0:
                    f2 -= 1;
                    printf("\nGroup %c attacked group %c!\n", g1,g2);
                    printf("Number of warriors for each group\nGroup %c: %i\nGroup %c: %i\n",g1,f1, g2,f2);
                break;
            case 1:
                    f1 -= 1;
                    printf("\nGroup %c attacked group %c!\n", g2,g1);
                    printf("Number of warriors for each group\nGroup %c: %i\nGroup %c: %i\n",g1,f1, g2,f2);
                break;
        }


    }
}

__global__ void monteCarlo(double m[3][3], int n, double *rand, int *attacker, int f0, int f1, int f2){
    double x;
    char zero='0', one='1', two='2';

    double newFactions[2][2] = {
            {0.0, 1.0},
            {1.0, 0.0},
    };

        if(f0 == 0){
            printf("\nGroup 0 is annihilated!\n");
            printf("==================================\nReconfiguring stochastic matrix\n");
            suddenDeath(newFactions,n, attacker,f1,f2,one, two);
        }else if(f1 == 0){
            printf("\nGroup 1 is annihilated!\n");
            printf("==================================\nReconfiguring stochastic matrix\n");
            suddenDeath(newFactions,n, attacker,f0,f2, zero,two);
        }else if(f2 == 0){
            printf("\nGroup 2 is annihilated!\n");
            printf("==================================\nReconfiguring stochastic matrix\n");
            suddenDeath(newFactions,n, attacker, f0,f1, zero, one);
        }

        switch(*attacker){
            case 0:
                x = m[0][1];

                if (*rand > 0.0 && *rand <= x){
                    f1 -= 1;
                    printf("\nGroup %i attacked group 1!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }else{
                    f2 -= 1;
                    printf("\nGroup %i attacked group 2!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }

                break;
            case 1:
                x = m[1][0];

                if (*rand > 0.0 && *rand <= x){
                    f0 -= 1;
                    printf("\nGroup %i attacked group 0!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }else{
                    f2 -= 1;
                    printf("\nGroup %i attacked group 2!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }

                break;
            case 2:
                x = m[2][0];

                if (*rand > 0.0 && *rand <= x){
                    f0 -= 1;
                    printf("\nGroup %i attacked group 0!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }else{
                    f1 -= 1;
                    printf("\nGroup %i attacked group 1!\n", *attacker);
                    printf("Number of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",f0, f1,f2);
                }

                break;
        }

}

double rands(double min, double max)
{
    double range = (max - min);
    double div = RAND_MAX / range;
    return min + (rand() / div);
}

int whoAttacks(int n, int min, int max){
    return n + rand() % (max + 1 - min) + 0;
}

void printMatrix(double m[3][3]) {
    printf("\n");
    for (int row = 0; row < 3; row++) {
        for (int column = 0; column < 3; column++) { printf("%.1f  ", m[row][column]); }
        printf("\n");
    }
}

int main(){
    int WA=5,WB=5,WC=5;

    //initial matrix
    double factions[3][3] = {
            {0.0, 0.7, 0.3},
            {0.4, 0.0, 0.6},
            {0.6, 0.4, 0.0}
    };

    printMatrix(factions);

    //initial warriors
    printf("\nNumber of warriors for each group\nGroup 0: %i\nGroup 1: %i\nGroup 2: %i\n",WA, WB,WC);
    printf("==================================\n");

    double size = sizeof(double);
    double *a, *rand;
    double *d_rand;
    int *attacker, *d_attacker;

    //reserve device memory
    hipMalloc((void**)&d_rand, size);
	hipMalloc((void**)&d_attacker, sizeof(int));

    //reserve host memory
    a = (double*)malloc(size);
    rand = (double*)malloc(size);
    attacker = (int*)malloc(sizeof(int));

    //things needed in order for the random numbers to work properly
    srand (time ( 0));
    *a = rands(0.0, 1.0);

    //MONTE CARLO FUNCTION
    for (int i = 0; i < N; ++i)
    {
        *attacker = whoAttacks(0,0,2);
        *rand = rands(0.0, 1.0);

        hipMemcpy(d_rand, rand, size, hipMemcpyHostToDevice);
	    hipMemcpy(d_attacker, attacker, size, hipMemcpyHostToDevice);

        monteCarlo << <N*N / THREADS_PER_BLOCK,  THREADS_PER_BLOCK >> >(factions, N, d_rand, d_attacker, WA,WB,WC);
    }
    
    //we clean the memory
    free(a);
    free(rand);
    free(attacker);

    return 0;
}
